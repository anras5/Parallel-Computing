#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <chrono>
#include <iomanip>

struct Row
{
    int *values;
    int numValues;

    __host__ __device__ Row() : values(nullptr), numValues(0) {}

    __host__ __device__ Row(int *vals, int count) : numValues(count)
    {
        values = new int[numValues];
        for (int i = 0; i < numValues; i++)
        {
            values[i] = vals[i];
        }
    }

    __host__ __device__ ~Row()
    {
        delete[] values;
    }
};

std::string *splitString(const std::string &line, char delimiter, int &numTokens)
{
    std::stringstream ss(line);
    std::string token;
    numTokens = 0;

    while (std::getline(ss, token, delimiter))
    {
        numTokens++;
    }

    std::string *tokens = new std::string[numTokens];
    ss.clear();
    ss.seekg(0);
    int i = 0;

    while (std::getline(ss, token, delimiter))
    {
        tokens[i++] = token;
    }

    return tokens;
}

Row *parseCSV(const std::string &filename, int &numRows)
{
    std::ifstream file(filename);
    std::string line;
    numRows = 0;

    if (!file.is_open())
    {
        std::cout << "Failed to open the file: " << filename << std::endl;
        return nullptr;
    }

    // Count the number of rows in the file
    while (std::getline(file, line))
    {
        numRows++;
    }

    // Allocate memory for rows
    Row *rows = new Row[numRows];
    file.clear();
    file.seekg(0);

    int i = 0;
    while (std::getline(file, line))
    {
        int numTokens;
        std::string *tokens = splitString(line, ',', numTokens);

        int *values = new int[numTokens];
        for (int j = 0; j < numTokens; j++)
        {
            values[j] = std::stoi(tokens[j]);
        }

        Row *newRow = new Row(values, numTokens);
        rows[i++] = *newRow;

        delete[] values;
        delete[] tokens;
    }

    file.close();

    return rows;
}

void printRows(Row *rows, int numRows)
{
    for (int i = 0; i < numRows; i++)
    {
        Row &row = rows[i];
        for (int j = 0; j < row.numValues; j++)
        {
            std::cout << row.values[j];
            if (j < row.numValues - 1)
            {
                std::cout << ",";
            }
        }
        std::cout << std::endl;
    }
}

__global__ void findAnomaliesKernel(Row *rows, int numRows, int *numAnomaliesArray, int *numCorrectArray)
{
    int threadNum = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = gridDim.x * blockDim.x;

    int numRowsPerThread = (numRows + numThreads - 1) / numThreads;
    int startRow = threadNum * numRowsPerThread;
    int endRow = min(startRow + numRowsPerThread, numRows);

    int numAnomalies = 0;
    int numCorrect = 0;

    for (int i = startRow; i < endRow; i++)
    {
        bool isConsistent = true;
        for (int j = 0; j < numRows; j++)
        {
            if (i != j)
            {
                bool isMatch = true;
                for (int k = 0; k < rows[i].numValues - 1; k++)
                {
                    if (rows[i].values[k] != rows[j].values[k])
                    {
                        isMatch = false;
                        break;
                    }
                }
                if (isMatch && rows[i].values[rows[i].numValues - 1] != rows[j].values[rows[j].numValues - 1])
                {
                    isConsistent = false;
                    break;
                }
            }
        }
        if (isConsistent)
        {
            numCorrect++;
        } else {
            numAnomalies++;
        }
    }

    numAnomaliesArray[threadNum] = numAnomalies;
    numCorrectArray[threadNum] = numCorrect;
}

void findAnomalies(Row *rows, int numRows, int &numAnomalies, int &numCorrect)
{
    hipError_t error;
    int numThreadsPerBlock = 256; // Choose an appropriate number of threads per block
    int numBlocks = (numRows + numThreadsPerBlock - 1) / numThreadsPerBlock;

    int *numAnomaliesArray;
    int *numCorrectArray;

    Row *devRows;
    hipMalloc((void **)&devRows, numRows * sizeof(Row));
    // Allocate memory for Row.values on the device
    for (int i = 0; i < numRows; i++)
    {
        int *devValues;
        hipMalloc((void **)&devValues, rows[i].numValues * sizeof(int));
        error = hipMemcpy(devValues, rows[i].values, rows[i].numValues * sizeof(int), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMemcpy (devValues,rows[i].values) returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
        rows[i].values = devValues;
    }

    error = hipMemcpy(devRows, rows, numRows * sizeof(Row), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("hipMemcpy (devRows,rows) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    hipMalloc((void **)&numAnomaliesArray, numBlocks * numThreadsPerBlock * sizeof(int));
    hipMalloc((void **)&numCorrectArray, numBlocks * numThreadsPerBlock * sizeof(int));

    findAnomaliesKernel<<<numBlocks, numThreadsPerBlock>>>(devRows, numRows, numAnomaliesArray, numCorrectArray);

    // Combine results from all threads
    numAnomalies = 0;
    numCorrect = 0;
    for (int i = 0; i < numBlocks * numThreadsPerBlock; i++)
    {
        int numAnomaliesThread = 0;
        error = hipMemcpy(&numAnomaliesThread, numAnomaliesArray + i, sizeof(int), hipMemcpyDeviceToHost);
        if (error != hipSuccess)
        {
            printf("hipMemcpy (numAnomaliesThread,numAnomaliesArray) returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
        numAnomalies += numAnomaliesThread;

        int numCorrectThread = 0;
        error = hipMemcpy(&numCorrectThread, numCorrectArray + i, sizeof(int), hipMemcpyDeviceToHost);
        if (error != hipSuccess)
        {
            printf("hipMemcpy (numCorrectThread,numCorrectArray) returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }
        numCorrect += numCorrectThread;
    }

    // Cleanup
    hipFree(devRows);
    hipFree(numAnomaliesArray);
    hipFree(numCorrectArray);
}

int main(int argc, char *argv[])
{
    if (argc < 2)
    {
        std::cout << "Usage: ./cuda_program <csv_filename>" << std::endl;
        return 1;
    }

    const char *filename = argv[1];
    int numRows;
    int numAnomalies = 0;
    int numCorrect = 0;
    Row *rows = parseCSV(filename, numRows);

    if (rows != nullptr)
    {
        auto start = std::chrono::high_resolution_clock::now();
        findAnomalies(rows, numRows, numAnomalies, numCorrect);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> duration = end - start;
        double executionTime = duration.count() * 1000;

        std::cout << numRows << ";" << std::fixed << std::setprecision(2) << executionTime << std::endl;

        // std::cout << "Number of rows in the file: " << numRows << std::endl;
        // std::cout << "Correct rows: " << numCorrect << std::endl;
        // if (numAnomalies == 0)
        // {
        //     std::cout << "Last column is dependent on the rest" << std::endl;
        // }
        // else
        // {
        //     std::cout << "Found anomalies: " << numAnomalies << std::endl;
        // }

        delete[] rows;
    }

    return 0;
}